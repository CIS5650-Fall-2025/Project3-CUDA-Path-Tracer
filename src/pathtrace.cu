#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/copy.h>
#include <thrust/gather.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "Light.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* dev_image, glm::vec3* dev_denoiseImg, glm::vec3* dev_final_image,
    float percentD)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    float percentRegular = 1 - percentD;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix1 = dev_image[index];
        glm::vec3 pix2 = dev_denoiseImg[index];
        glm::vec3 pix = percentRegular * pix1 + percentD * pix2;

        dev_final_image[index] = pix;

        glm::ivec3 color;

        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static glm::vec3* dev_denoiseImg = NULL;
static glm::vec3* dev_final_image = NULL;
static glm::vec3* dev_normalsImg = NULL;
static glm::vec3* dev_albedoImg = NULL;

static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static MeshTriangle* dev_triangleBuffer_0 = NULL;

static std::vector<hipTextureObject_t> host_texObjs;
static std::vector<hipArray_t> dev_cuArrays;
static hipTextureObject_t* dev_textureObjIDs;
static BVHNode* dev_bvhNodes = NULL;

static AreaLight* dev_areaLights = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_denoiseImg, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoiseImg, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_final_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_final_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normalsImg, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normalsImg, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_albedoImg, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedoImg, 1, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    //Initialize Triangle Memory!
    std::vector<MeshTriangle>* triangles = hst_scene->getTriangleBuffer();
    //std::cout << "# of triangles: " << triangles->size() << "\n";

    if (triangles != nullptr) {

        hipMalloc(&dev_triangleBuffer_0, (*triangles).size() * sizeof(MeshTriangle));
        hipMemcpy(dev_triangleBuffer_0, (*triangles).data(), triangles->size() * sizeof(MeshTriangle), hipMemcpyHostToDevice);
        checkCUDAError("Triangle Buffer Init");

        /// CUDA TEXTURE OBJECTS!
        std::vector<tinygltf::Image> images = hst_scene->getImages();
        for (const tinygltf::Image& image : images) {
            hipChannelFormatKind formatType;
            hipChannelFormatDesc channelDesc;
            if (image.pixel_type == TINYGLTF_COMPONENT_TYPE_UNSIGNED_BYTE) {
                formatType = hipChannelFormatKindUnsigned;
                if (image.component == 3) {
                    channelDesc = hipCreateChannelDesc<uchar3>();
                }
                else {
                    channelDesc = hipCreateChannelDesc<uchar4>();
                }
            }
            else {
                formatType = hipChannelFormatKindFloat;
                if (image.component == 3) {
                    channelDesc = hipCreateChannelDesc<float3>();
                }
                else {
                    channelDesc = hipCreateChannelDesc<float4>();
                }
            }
            hipArray_t cuArray;
            hipMallocArray(&cuArray, &channelDesc, image.width, image.height);
            hipMemcpy2DToArray(cuArray, 0, 0,
                image.image.data(),
                image.width * sizeof(float),
                image.width * sizeof(float),
                image.height,
                hipMemcpyHostToDevice);
            //checkCUDAError("aight");
            dev_cuArrays.push_back(cuArray);

            //// Specify texture
            struct hipResourceDesc resDesc;
            memset(&resDesc, 0, sizeof(resDesc));
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = cuArray;

            // Specify texture object parameters
            struct hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.addressMode[0] = hipAddressModeWrap;
            texDesc.addressMode[1] = hipAddressModeWrap;
            texDesc.filterMode = hipFilterModePoint;
            texDesc.readMode = hipReadModeElementType;
            texDesc.normalizedCoords = 1;

            hipTextureObject_t texObj = 0;
            hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
            checkCUDAError("textureObject Init");
            host_texObjs.push_back(texObj);
        }

        hipMalloc((void**)&dev_textureObjIDs, host_texObjs.size() * sizeof(hipTextureObject_t));
        hipMemcpy(dev_textureObjIDs, host_texObjs.data(), host_texObjs.size() * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
        checkCUDAError("images init");

        /// BVH TREE
        std::vector<BVHNode> nodes = hst_scene->getBvhNode();
        hipMalloc(&dev_bvhNodes, nodes.size() * sizeof(BVHNode));
        hipMemcpy(dev_bvhNodes, nodes.data(), nodes.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
        checkCUDAError("BVH tree init");
    }
    else {
        std::cout << "No triangles!\n";
    }

    //Initialize Light Device Memory!

    hipMalloc(&dev_areaLights, scene->areaLights.size() * sizeof(AreaLight));
    hipMemcpy(dev_areaLights, scene->areaLights.data(), scene->areaLights.size() * sizeof(AreaLight), hipMemcpyHostToDevice);
    //std::cout << "all cuda mem initialized!\n";
    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_denoiseImg);
    hipFree(dev_final_image);
    hipFree(dev_normalsImg);
    hipFree(dev_albedoImg);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_triangleBuffer_0);

    for (hipArray_t cuArray : dev_cuArrays) {
        if (cuArray != nullptr) {
            hipError_t err = hipFreeArray(cuArray);
            if (err != hipSuccess) {
                std::cerr << "Failed to free CUDA array: " << hipGetErrorString(err) << std::endl;
            }
        }
    }
    dev_cuArrays.clear();

    for (int i = 0; i < host_texObjs.size(); i++) {
        hipTextureObject_t texObj = host_texObjs[i];
        hipError_t err = hipDestroyTextureObject(texObj);
        if (err != hipSuccess) {
            std::cerr << "Failed to destroy texture object: " << hipGetErrorString(err) << std::endl;
        }
        hipDestroyTextureObject(host_texObjs[i]);
    }
    host_texObjs.clear();


    hipFree(dev_textureObjIDs);

    hipFree(dev_bvhNodes);

    hipFree(dev_areaLights);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.L = glm::vec3(0.0f, 0.0f, 0.0f); // Used to be (1.0, 1.0, 1.0)
        segment.beta = glm::vec3(1, 1, 1);

/// ANTI ALIASING
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> uhalf(0.0, 0.5);

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + 0.5f + uhalf(rng))
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + 0.5f + uhalf(rng))
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
        segment.lastHitWasSpecular = false;
    }
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    AreaLight* areaLights,
    MeshTriangle* triangles,
    hipTextureObject_t* texObjs,
    BVHNode* bvhNodes,
    bool BVHEmpty,
    int geoms_size,
    int num_areaLights,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    //Don't compute if the segment is already complete!
    if (path_index < num_paths && pathSegments[path_index].remainingBounces > 0)
    {

        //intersections[path_index].t = 3;
        PathSegment pathSegment = pathSegments[path_index];

        //float t;
        //glm::vec3 intersect_point;
        //glm::vec3 normal;
        //glm::vec3 texCol;
        float t_min = FLT_MAX;
        intersections[path_index].t = -1;
        intersections[path_index].materialId = -1;
        intersections[path_index].areaLightId = -1;
        //// 1. BVH for all triangles
        if (!BVHEmpty) {
            BVHIntersect(pathSegment.ray, intersections[path_index], triangles, bvhNodes, texObjs);
            if (intersections[path_index].t != -1) {
                t_min = intersections[path_index].t;
            }
        }

        bool hitLight = AllLightIntersectTest(intersections[path_index], pathSegment.ray,
            triangles, bvhNodes,
            areaLights, num_areaLights);

        t_min = intersections[path_index].t == -1 ? intersections[path_index].t : FLT_MAX;
        float t = -1;
        int matID = -1;
        glm::vec3 normal = glm::vec3(0);
        for (int i = 0; i < geoms_size; i++) {
            Geom& geom = geoms[i];
            glm::vec3 tmp_normal;
            glm::vec3 tmp_intersect;
            bool outside = true;
            
            if (geom.type == G_SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }

            if (t > 0.0f && t < t_min)
            {
                t_min = t;
                matID = geom.materialid;
                normal = tmp_normal;
            }
        }
        if (matID != -1) {
            intersections[path_index].t = t_min;
            intersections[path_index].areaLightId = -1;
            intersections[path_index].materialId = matID;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].texCol = glm::vec3(-1);
        }
    }
}

/**
* Accumulate normals and albedo in their buffers
*/
__global__ void denoise_shade(
    int num_paths,
    AreaLight* areaLights,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    glm::vec3* normalsImg,
    glm::vec3* albedoImg,
    int curItr,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        int pixelIndex = pathSegments[idx].pixelIndex;
        if (intersection.t > 0) { //intersection
            normalsImg[pixelIndex] *= (curItr - 1);
            normalsImg[pixelIndex] += intersection.surfaceNormal;
            normalsImg[pixelIndex] /= curItr;
            if (intersection.areaLightId != -1) {
                glm::vec3 a = areaLights[intersection.areaLightId].Le * areaLights[intersection.areaLightId].emittance;
                a = glm::clamp(a, glm::vec3(0), glm::vec3(1));
                albedoImg[pixelIndex] *= (curItr - 1);
                albedoImg[pixelIndex] += a;
                albedoImg[pixelIndex] /= curItr;
            }
            else {
                Material material = materials[intersection.materialId]; //In BVH intersection, I guarantee that materialId must be valid if t > 0
                glm::vec3 color = (intersection.texCol.x != -1) ? intersection.texCol : material.color;
                glm::vec3 a = glm::clamp(color, glm::vec3(0), glm::vec3(1));
                albedoImg[pixelIndex] *= (curItr - 1);
                albedoImg[pixelIndex] += a;
                albedoImg[pixelIndex] /= curItr;
            }
//DEPTH TESTING
            //albedoImg[pixelIndex] = glm::vec3(intersection.t / 20.0f);
            //albedoImg[pixelIndex] = glm::vec3(1, 0, 1);
//DEPTH TESTING
        }
        else { //no intersection
            float factor = ((curItr - 1) / curItr);
            normalsImg[pixelIndex] *= factor;
            albedoImg[pixelIndex] *= factor;
        }
    }
}

__global__ void full_lighting_shade(int traceDepth, int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    MeshTriangle* triangles,
    BVHNode* bvhNodes,
    AreaLight* areaLights,
    hipTextureObject_t* texObjs,
    int num_areaLights,
    const bool BVHEmpty)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        bool useTexCol = (intersection.texCol.x != -1);
        if (intersection.t > 0 && pathSegments[idx].remainingBounces > 0) {

            
            if (intersection.areaLightId != -1) {
                if (pathSegments[idx].lastHitWasSpecular || pathSegments[idx].remainingBounces == traceDepth) {
                    pathSegments[idx].L += areaLights[intersection.areaLightId].Le * areaLights[intersection.areaLightId].emittance
                        * pathSegments[idx].beta;
                    pathSegments[idx].remainingBounces = 0;
                    return;
                }
            }
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
            Material material = materials[intersection.materialId];
            pathSegments[idx].remainingBounces--;
            pathSegments[idx].ray.origin = getPointOnRay(pathSegments[idx].ray, intersection.t);
            MatType mt = material.type;


            float pdf = 0;
            glm::vec3 f =  glm::vec3(0);
            glm::vec3 woWOut = -pathSegments[idx].ray.direction;

            if (mt == SPEC_REFL || mt == SPEC_TRANS || mt == SPEC_GLASS || mt == DIAMOND) {
                sample_f(pathSegments[idx], woWOut, pdf, f, intersection.surfaceNormal, material, intersection.texCol, useTexCol, rng);

                if (pdf < 0.0000001f || f == glm::vec3(0))
                {
                    pathSegments[idx].remainingBounces = 0;
                    return;
                }

                float absdot = glm::abs(glm::dot(pathSegments[idx].ray.direction, intersection.surfaceNormal));
                pathSegments[idx].beta *= f * absdot / pdf;
                pathSegments[idx].lastHitWasSpecular = true;
            }
            else {
                glm::vec3 viewPoint = pathSegments[idx].ray.origin;
                glm::vec3 direct_L = MISDirectLi(triangles, bvhNodes, areaLights, texObjs,
                    num_areaLights,
                    woWOut, viewPoint, intersection.surfaceNormal,
                    material, intersection.texCol, useTexCol,
                    BVHEmpty,
                    rng);

                pathSegments[idx].L += pathSegments[idx].beta * direct_L;
                sample_f(pathSegments[idx], woWOut, pdf, f, intersection.surfaceNormal, material, intersection.texCol, useTexCol, rng);

                if (pdf < 0.0000001f || f == glm::vec3(0))
                {
                    pathSegments[idx].remainingBounces = 0;
                    return;
                }

                float absdot = glm::abs(glm::dot(pathSegments[idx].ray.direction, intersection.surfaceNormal));
                pathSegments[idx].beta *= f * absdot / pdf;
                pathSegments[idx].lastHitWasSpecular = false;
            }
        }
        else {
            pathSegments[idx].remainingBounces = 0;
            return;
        }
    }
}





__global__ void simple_direct_shade(int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    MeshTriangle* triangles,
    BVHNode* bvhNodes,
    AreaLight* areaLights)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        bool useTexCol = (intersection.texCol.x != -1);
        if (intersection.t > 0 && pathSegments[idx].remainingBounces > 0) {
            if (intersection.areaLightId != -1) {
//HIT A LIGHT: NAIVE EXIT CASE
                pathSegments[idx].L = areaLights[intersection.areaLightId].Le * areaLights[intersection.areaLightId].emittance;
                pathSegments[idx].remainingBounces = 0;
                return;
            }

            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
            Material material = materials[intersection.materialId];

            pathSegments[idx].ray.origin = getPointOnRay(pathSegments[idx].ray, intersection.t);
            pathSegments[idx].remainingBounces--;

            //TODO
            //SIMPLE 1 BOUNCE DIRECTLIGHT LI
            float pdf;
            glm::vec3 wiW;

            int chosenLightIdx, chosenLightID;
            LightType chosenLightType;

            glm::vec3 Li = Sample_Li(triangles, bvhNodes, areaLights, 1, pathSegments[idx].ray.origin,
                intersection.surfaceNormal,
                wiW, pdf, chosenLightIdx, chosenLightID, chosenLightType, rng);

            //TEST
            //pathSegments[idx].L = 0.5f * (wiW + glm::vec3(1.));
            //pathSegments[idx].remainingBounces = 0;
            //return;
            //TEST
            if (pdf == 0) {
                return;
            }

            glm::vec3 f_col;
            glm::vec3 woWOut = -pathSegments[idx].ray.direction;
            f(woWOut, wiW, pdf, f_col, intersection.surfaceNormal, material, intersection.texCol, useTexCol, rng);
            pathSegments[idx].L = f_col * Li * abs(dot(wiW, intersection.surfaceNormal)) / pdf;
            pathSegments[idx].ray.direction = wiW;
            pathSegments[idx].remainingBounces = 0;
            return;
        }
        else {
            return;
        }
    }
}





///  Iterative lighting logic:
///  LTE:
///  L_o = L_e + integral(f() * Li(w_i) * absdot)_dw_i
///  L_o = L_e + (f() * Li(w_i) * absdot) / pdf(w_i)
__global__ void naive_shade(int iter,
    int num_paths,
    int traceDepth,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    MeshTriangle* triangles,
    BVHNode* bvhNodes,
    AreaLight* areaLights)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        bool useTexCol = (intersection.texCol.x != -1);
        if (intersection.t > 0 && pathSegments[idx].remainingBounces > 0) {
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
            Material material = materials[intersection.materialId];

            pathSegments[idx].ray.origin = getPointOnRay(pathSegments[idx].ray, intersection.t);
            pathSegments[idx].remainingBounces--;

            //if (pathSegments[idx].remainingBounces == traceDepth - 1 && material.type == MATTEBLACK) {
            //    pathSegments[idx].L = material.color;
            //    pathSegments[idx].remainingBounces = 0;
            //    return;
            //}

            if (intersection.areaLightId != -1) {
                //HIT A LIGHT: NAIVE EXIT CASE
                glm::vec3 Le = areaLights[intersection.areaLightId].Le * areaLights[intersection.areaLightId].emittance;
                pathSegments[idx].L = pathSegments[idx].beta * Le;
                //pathSegments[idx].L = glm::clamp(pathSegments[idx].L, glm::vec3(0), Le);
                pathSegments[idx].remainingBounces = 0;
                return;
            }
            
            float pdf = 0;
            glm::vec3 f = glm::vec3(0);
            glm::vec3 woWOut = -pathSegments[idx].ray.direction;
            sample_f(pathSegments[idx], woWOut, pdf, f, intersection.surfaceNormal, material, intersection.texCol, useTexCol, rng);

            if (pdf < 0.0000001f || f == glm::vec3(0))
            {
                pathSegments[idx].remainingBounces = 0;
                return;
            }

            float absdot = glm::abs(glm::dot(pathSegments[idx].ray.direction, intersection.surfaceNormal));
            pathSegments[idx].beta *= f * absdot / pdf;
        }
        else {
            pathSegments[idx].remainingBounces = 0;
            return;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths, int cur_iter)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] *= (cur_iter - 1);
        image[iterationPath.pixelIndex] += iterationPath.L; //should be L, not beta
        image[iterationPath.pixelIndex] /= cur_iter;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, oidn::FilterRef& oidn_filter, float& percentD, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    //const int traceDepth = 100;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////
    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");
    
    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
/// Clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

/// TRACE 1 DEPTH (COMPUTE INTERSECTIONS)
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            dev_areaLights,
            dev_triangleBuffer_0,
            dev_textureObjIDs,
            dev_bvhNodes,
            hst_scene->isBVHEmpty,
            hst_scene->geoms.size(),
            hst_scene->areaLights.size(),
            dev_intersections
        );

        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

/// ALBEDO AND NORMAL BUFFERS
        //For every iteration, at the first intersection!
        if (depth == 1) {
            denoise_shade<<<numblocksPathSegmentTracing, blockSize1d>>>(
                num_paths,
                dev_areaLights,
                dev_intersections,
                dev_paths,
                dev_normalsImg,
                dev_albedoImg,
                iter,
                dev_materials
            );
        }
        
/// TOGGLEABLE: SORT BY MATERIAL OPTIMIZATION
        if (guiData != NULL && guiData->SortByMat)
        {
            thrust::device_ptr<ShadeableIntersection> d_itr_ptr(dev_intersections);
            thrust::device_ptr<PathSegment> d_paths_ptr(dev_paths);
            thrust::device_vector<int> d_keys(num_paths);
            thrust::transform(d_itr_ptr, d_itr_ptr + num_paths, d_keys.begin(), getMatId());

            //sort both d_itr_ptr and d_paths_ptr based on the sorting of the materialID buffer
            thrust::sort_by_key(d_keys.begin(), d_keys.begin() + num_paths,
                thrust::make_zip_iterator(thrust::make_tuple(d_itr_ptr, d_paths_ptr)));
        }

        /// SHADING

        //full_lighting_shade<<<numblocksPathSegmentTracing, blockSize1d>>>(
        //    traceDepth,
        //    iter,
        //    num_paths,
        //    dev_intersections,
        //    dev_paths,
        //    dev_materials,
        //    dev_triangleBuffer_0,
        //    dev_bvhNodes,
        //    dev_areaLights,
        //    dev_textureObjIDs,
        //    hst_scene->areaLights.size(),
        //    hst_scene->isBVHEmpty
        //);

        //simple_direct_shade<<<numblocksPathSegmentTracing, blockSize1d>>>(
        //    iter,
        //    num_paths,
        //    dev_intersections,
        //    dev_paths,
        //    dev_materials,
        //    dev_triangleBuffer_0,
        //    dev_bvhNodes,
        //    dev_areaLights
        //    );

        naive_shade<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            traceDepth,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_triangleBuffer_0,
            dev_bvhNodes,
            dev_areaLights
        );

        checkCUDAError("shade 1 depth of path segments");

/// TOGGLEABLE: STREAM COMPACTION OPTIMIZATION
        if (guiData != NULL && guiData->StreamCompaction)
        {
            thrust::device_ptr<PathSegment> d_ptr(dev_paths);

            auto new_end = thrust::partition(d_ptr, d_ptr + num_paths, CheckRemainingBounces());

            num_paths = thrust::distance(d_ptr, new_end);
        }

        if (num_paths == 0 || depth >= traceDepth) {
            iterationComplete = true;
        }
        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }
    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths, iter);
    checkCUDAError("finalGather step on beauty pass (dev_image)");

    // Run denoising!
    //
    if (iter % 10 == 0) {
        oidn_filter.setImage("color", dev_image, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
        oidn_filter.setImage("albedo", dev_albedoImg, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
        oidn_filter.setImage("normal", dev_normalsImg, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);
        oidn_filter.setImage("output", dev_denoiseImg, oidn::Format::Float3, cam.resolution.x, cam.resolution.y);

        oidn_filter.commit();
        oidn_filter.execute();
    }

    ///////////////////////////////////////////////////////////////////////////
    // Send results to OpenGL buffer for rendering
    // Modify this to send dev_denoiseImg instead of dev_image!
    
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image, dev_denoiseImg, dev_final_image, guiData->PercentDenoise);
    
    // Retrieve image from GPU
    if (iter % 10 == 0) {
        hipMemcpy(hst_scene->state.image.data(), dev_final_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    }

    checkCUDAError("pathtrace");
}
