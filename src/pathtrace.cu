#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <OpenImageDenoise/oidn.hpp>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static Triangle* dev_triangles = NULL;
static BVHNode* dev_bvhnodes = NULL;
static glm::vec4* dev_tex_data = NULL;
static glm::vec4* dev_bumpmap_data = NULL;
static int* dev_tex_starts = NULL;
static int* dev_bump_starts = NULL;
static glm::vec2* dev_tex_dims = NULL;
static glm::vec2* dev_bump_dims = NULL;
static glm::vec4* dev_environmentmap_data = NULL;
static glm::vec2* dev_environmentmap_dim = NULL;

static oidn::DeviceRef oidn_device;
static glm::vec3* dev_oidn_normal = NULL;
static glm::vec3* dev_oidn_normalized_normal = NULL;
static glm::vec3* dev_oidn_albedo = NULL;
static glm::vec3* dev_oidn_normalized_albedo = NULL;
static glm::vec3* dev_oidn_filtered_image = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    if (scene->meshes.size() > 0) {
        int num_tris = scene->triangle_count;
        hipMalloc(&dev_triangles, num_tris * sizeof(Triangle));
        hipMemcpy(dev_triangles, scene->mesh_triangles.data(), num_tris * sizeof(Triangle), hipMemcpyHostToDevice);

        int num_bvhnodes = scene->bvhNodes.size();
        hipMalloc(&dev_bvhnodes, num_bvhnodes * sizeof(BVHNode));
        hipMemcpy(dev_bvhnodes, scene->bvhNodes.data(), num_bvhnodes * sizeof(BVHNode), hipMemcpyHostToDevice);
    }

    if (scene->textures.size() > 0) {
        int num_colors = 0;
        std::vector<glm::vec4> all_colors;
        for (Texture& tex : scene->textures) {
            num_colors += tex.color_data.size();
            all_colors.insert(all_colors.end(), tex.color_data.begin(), tex.color_data.end());
        }
        hipMalloc(&dev_tex_data, num_colors * sizeof(glm::vec4));
        hipMemcpy(dev_tex_data, all_colors.data(), num_colors * sizeof(glm::vec4), hipMemcpyHostToDevice);

        hipMalloc(&dev_tex_starts, scene->tex_starts.size() * sizeof(int));
        hipMemcpy(dev_tex_starts, scene->tex_starts.data(), scene->tex_starts.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dev_tex_dims, scene->tex_dims.size() * sizeof(glm::vec2));
        hipMemcpy(dev_tex_dims, scene->tex_dims.data(), scene->tex_dims.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    if (scene->bumpmaps.size() > 0) {
        int num_normals = 0;
        std::vector<glm::vec4> all_normals;
        for (Texture& tex : scene->bumpmaps) {
            num_normals += tex.color_data.size();
            all_normals.insert(all_normals.end(), tex.color_data.begin(), tex.color_data.end());
        }
        hipMalloc(&dev_bumpmap_data, num_normals * sizeof(glm::vec4));
        hipMemcpy(dev_bumpmap_data, all_normals.data(), num_normals * sizeof(glm::vec4), hipMemcpyHostToDevice);

        hipMalloc(&dev_bump_starts, scene->bump_starts.size() * sizeof(int));
        hipMemcpy(dev_bump_starts, scene->bump_starts.data(), scene->bump_starts.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dev_bump_dims, scene->bump_dims.size() * sizeof(glm::vec2));
        hipMemcpy(dev_bump_dims, scene->bump_dims.data(), scene->bump_dims.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);
    }

    //pass this regardless of if environmentmap is enabled so it can be used to check for environment map usage
    hipMalloc(&dev_environmentmap_dim, sizeof(glm::vec2));
    hipMemcpy(dev_environmentmap_dim, &(scene->environmentmap_dim), sizeof(glm::vec2), hipMemcpyHostToDevice);
    if (scene->environmentmap) {
        hipMalloc(&dev_environmentmap_data, scene->environmentmap->color_data.size() * sizeof(glm::vec4));
        hipMemcpy(dev_environmentmap_data, scene->environmentmap->color_data.data(), scene->environmentmap->color_data.size() * sizeof(glm::vec4), hipMemcpyHostToDevice);
    }

    checkCUDAError("pathtraceInit");

    //set up oidn and bufs
    oidn_device = oidnNewDevice(OIDNDeviceType::OIDN_DEVICE_TYPE_CUDA);
    oidn_device.commit();

    hipMalloc(&dev_oidn_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_oidn_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_oidn_normalized_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_oidn_normalized_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_oidn_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_oidn_normal, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_oidn_normalized_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_oidn_normalized_normal, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_oidn_filtered_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_oidn_filtered_image, 0, pixelcount * sizeof(glm::vec3));
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_triangles);
    hipFree(dev_bvhnodes);
    hipFree(dev_tex_data);
    hipFree(dev_bumpmap_data);
    hipFree(dev_environmentmap_data);
    hipFree(dev_environmentmap_dim);
    hipFree(dev_tex_starts);
    hipFree(dev_bump_starts);
    hipFree(dev_tex_dims);
    hipFree(dev_bump_dims);

    hipFree(dev_oidn_albedo);
    hipFree(dev_oidn_normalized_albedo);
    hipFree(dev_oidn_normal);
    hipFree(dev_oidn_normalized_normal);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0, 1);

#define USE_ANTIALIASING 1
#if USE_ANTIALIASING
        // antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
        );
#else
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * (x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * (y - (float)cam.resolution.y * 0.5f)
        );
#endif

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    Triangle* tris,
    int num_tris,
    BVHNode* bvhnodes)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        bool outside = true;
        int material_tex_id{ -1 };
        int bumpmap_id{ -1 };

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        int tmp_material_tex_id{ -1 };
        int tmp_bumpmap_id{ -1 };

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                tmp_material_tex_id = geom.materialid;
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                intersections->outside = outside;
                tmp_material_tex_id = geom.materialid;
            }
            else if (geom.type == TRIANGLE) {
                //t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
                material_tex_id = tmp_material_tex_id;
                bumpmap_id = tmp_bumpmap_id;
            }
        }

        //mesh intersection
        if (num_tris > 0) {
#define USE_BVH 1
#if USE_BVH
            t = bvhIntersectionTest(pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tmp_material_tex_id, tmp_bumpmap_id, outside, bvhnodes, tris, num_tris);
#else
            t = naiveMeshIntersectionTest(pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tmp_material_tex_id, tmp_bumpmap_id, outside, tris, num_tris);
#endif
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
                material_tex_id = tmp_material_tex_id;
                bumpmap_id = tmp_bumpmap_id;
            }
        }

        intersections[path_index].outside = outside;

        if (material_tex_id  == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = material_tex_id;
            intersections[path_index].bumpmapId = bumpmap_id;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uv = uv;
        }
    }
}

__device__ float fresnelDielectricEval(float etaI, float etaT, float cosThetaI, float cosThetaT) {
    float rhoParallel = ((etaT * cosThetaI) - (etaT * cosThetaT)) / ((etaT * cosThetaI) + (etaT * cosThetaT));
    float rhoPerp = ((etaI * cosThetaI) - (etaT * cosThetaT)) / ((etaI * cosThetaI) + (etaT * cosThetaT));
    return (rhoParallel * rhoParallel + rhoPerp * rhoPerp) * 0.5f;
}

__device__ bool refract(glm::vec3& rd, glm::vec3& nor, glm::vec3& refracted, float eta, float& cosTheta) {
    glm::vec3 normalized_dir = glm::normalize(rd);
    float dot_res = glm::dot(normalized_dir, nor);
    float discrim = 1.f - eta * eta * (1.f - dot_res * dot_res);
    if (discrim != 0.f) {
        cosTheta = sqrt(discrim);
        refracted = eta * (normalized_dir - nor * dot_res) - nor * cosTheta;
        return true;
    }
    return false;
}

__device__ float cosTheta(glm::vec3 v1, glm::vec3 v2) {
    return glm::cos(glm::acos(glm::dot(v1, v2)));
}

__device__ float random(glm::vec2 in) {
    return glm::fract(glm::sin(glm::dot(glm::vec2(in.x, in.y),
        glm::vec2(4.1235, 214.21))) *
        214125.123);
}

__global__ void shadeMaterials(int iter,
                               int num_paths,
                               int depth,
                               ShadeableIntersection* shadeableIntersections,
                               PathSegment* pathSegments,
                               Material* materials,
                               glm::vec4* texture_data,
                               glm::vec4* bumpmap_data,
                               glm::vec4* environmentmap_data,
                               glm::vec2* environmentmap_dim,
                               int* tex_starts,
                               int* bump_starts,
                               glm::vec2* tex_dims,
                               glm::vec2* bump_dims,
                               glm::vec3* oidn_albedo,
                               glm::vec3* oidn_normals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx > num_paths || pathSegments[idx].remainingBounces <= 0) {
        return;
    }

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) // if the intersection exists...
    {
        // Set up the RNG
        // LOOK: this is how you use thrust's RNG! Please look at
        // makeSeededRandomEngine as well.
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
        thrust::uniform_real_distribution<float> u01(0, 1);

        PathSegment& curr_seg = pathSegments[idx];
        Ray& curr_ray = curr_seg.ray;

        Material material = materials[intersection.materialId];
        glm::vec3 materialColor = material.color;


#define USE_BUMP_MAP 1
#if USE_BUMP_MAP
        if (intersection.bumpmapId != -1) {
            int bumpmap_idx = materials[intersection.bumpmapId].bumpmap_index;
            int start_idx = bump_starts[bumpmap_idx];
            glm::vec2 dims = bump_dims[bumpmap_idx];

            glm::vec2& uv = intersection.uv;
            float tex_x = glm::fract(uv.x) * dims.x;
            float tex_y = glm::fract(1.0f - uv.y) * dims.y;
            float x_u = glm::fract(tex_x), y_u = glm::fract(tex_y);
            float x_f = glm::floor(tex_x), y_f = glm::floor(tex_y);
            float x_c = glm::ceil(tex_x), y_c = glm::ceil(tex_y);

            int bl_idx = start_idx + y_f * dims.x + x_f;
            int br_idx = start_idx + y_f * dims.x + x_c;
            int tl_idx = start_idx + y_c * dims.x + x_f;
            int tr_idx = start_idx + y_c * dims.x + x_c;

            glm::vec4 bl_tex = bumpmap_data[bl_idx];
            glm::vec4 br_tex = bumpmap_data[br_idx];
            glm::vec4 tl_tex = bumpmap_data[tl_idx];
            glm::vec4 tr_tex = bumpmap_data[tr_idx];

            glm::vec4 tex_val = glm::mix(glm::mix(bl_tex, tl_tex, y_u), glm::mix(br_tex, tr_tex, y_u), x_u);

            glm::vec3 tangent = glm::cross(intersection.surfaceNormal, curr_ray.direction);
            glm::vec3 bitangent = glm::cross(intersection.surfaceNormal, tangent);
            glm::mat3 nor_transform{ glm::normalize(tangent), glm::normalize(bitangent), glm::normalize(intersection.surfaceNormal) };

            intersection.surfaceNormal = glm::normalize(nor_transform * glm::vec3(tex_val));
        }
#endif
        
#define USE_TEXTURE 1
#if USE_TEXTURE
        if (material.tex_index != -1) {
            int tex_index = material.tex_index;
            int start_idx = tex_starts[tex_index];
            glm::vec2& dims = tex_dims[tex_index];
            glm::vec2& uv = intersection.uv;

            int tex_x_idx = glm::fract(uv.x) * dims.x;
            int tex_y_idx = glm::fract(1.0f - uv.y) * dims.y;
            int tex_1d_idx = start_idx + tex_y_idx * dims.x + tex_x_idx;

#define USE_PROCEDURAL_TEXTURE 0
#if !USE_PROCEDURAL_TEXTURE
            materialColor = glm::vec3(texture_data[tex_1d_idx]);
#else
            glm::vec3 p = glm::vec3((glm::vec2(tex_x_idx, tex_y_idx) - .5f * dims) / dims.y, floor(glm::mod(random(uv) * 3146.f, 8.0f)));
            materialColor = glm::vec3(glm::pow(matWood(p), glm::vec3(.4545f)));
#endif
        }
#endif

        //now that we have modified nor + albedo we can update the oidn bufs
        if (depth == 1) {
            oidn_albedo[idx] += materialColor;
            oidn_normals[idx] += intersection.surfaceNormal;
        }

        // If the material indicates that the object was a light, "light" the ray
        if (material.emittance > 0.0f) {
            curr_seg.color *= (materialColor * material.emittance);
            curr_seg.remainingBounces = 0;
        } 
        else  if (material.specular_transmissive.isSpecular == false) {
            //perfectly diffuse for now
            glm::vec3 nor = intersection.surfaceNormal;
            glm::vec3 isect_pt = glm::normalize(curr_ray.direction) * intersection.t + curr_ray.origin;

            glm::vec3 wi;
            scatterRay(curr_seg, isect_pt, intersection.surfaceNormal, material, rng, wi);

            wi = glm::normalize(wi);

            float costheta = cosTheta(wi, intersection.surfaceNormal);
            float pdf = costheta * INV_PI;
            if (pdf == 0.f) {
                curr_seg.remainingBounces = 0;
                return;
            }

            glm::vec3 bsdf = materialColor * INV_PI;
            float lambert = glm::abs(glm::dot(wi, intersection.surfaceNormal));

            curr_seg.color *= (bsdf * lambert) / pdf;

            glm::vec3 new_dir = wi;
            glm::vec3 new_origin = isect_pt + intersection.surfaceNormal * 0.01f;
            curr_seg.ray.origin = new_origin;
            curr_seg.ray.direction = new_dir;
            curr_seg.remainingBounces--;
        }
        else if (material.specular_transmissive.isSpecular == true && material.specular_transmissive.isTransmissive == false) {
            //perfectly specular
            glm::vec3 nor = intersection.surfaceNormal;
            glm::vec3 isect_pt = glm::normalize(curr_ray.direction) * intersection.t + curr_ray.origin;

            glm::vec3 wi = glm::reflect(curr_ray.direction, intersection.surfaceNormal);

            wi = glm::normalize(wi);

            //took out lambert and INV_PI from bsdf
            glm::vec3 bsdf = materialColor;
            float lambert = glm::abs(glm::dot(wi, intersection.surfaceNormal));

            curr_seg.color *= (bsdf); //pdf = 1

            glm::vec3 new_dir = wi;
            glm::vec3 new_origin = isect_pt + intersection.surfaceNormal * 0.01f;
            curr_seg.ray.origin = new_origin;
            curr_seg.ray.direction = new_dir;
            curr_seg.remainingBounces--;
        }
        else if (material.specular_transmissive.isSpecular == true && material.specular_transmissive.isTransmissive == true) {
            
            glm::vec3 nor = intersection.surfaceNormal;
            glm::vec3 isect_pt = glm::normalize(curr_ray.direction) * intersection.t + curr_ray.origin;

            float rand_num = u01(rng);

            glm::vec3 wi, bsdf;

            float etaA = material.specular_transmissive.eta.x;
            float etaB = material.specular_transmissive.eta.y;
            float etaI, etaT;
            
            if (glm::dot(curr_ray.direction, intersection.surfaceNormal) > 0.f) {
                etaI = etaB;
                etaT = etaA;
                nor = -nor;
            }
            else {
                etaI = etaA;
                etaT = etaB;
            }

            float eta = etaI / etaT;

            glm::vec3 rd = curr_ray.direction;
            float cosThetaI = glm::dot(rd, -nor) / glm::length(rd);
            float cosThetaT;
            glm::vec3 refracted, reflected = glm::reflect(rd, nor);

            bool can_refract{ true };

            if (!refract(rd, nor, refracted, eta, cosThetaT)) {
                can_refract = false;
            }

            float fresnel = fresnelDielectricEval(etaI, etaT, cosThetaI, cosThetaT);

            if (rand_num < fresnel || !can_refract) {
                // Reflect
                curr_ray.direction = reflected;
                curr_ray.origin = isect_pt + reflected * 0.01f;
            }
            else {
                // Refract
                curr_ray.direction = refracted;
                curr_ray.origin = isect_pt + refracted * 0.01f;
            }

            bsdf = materialColor;

            curr_seg.color *= (bsdf); //pdf = 1
            curr_seg.remainingBounces--;
        }
        // If there was no intersection, color the ray black.
        // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
        // used for opacity, in which case they can indicate "no opacity".
        // This can be useful for post-processing and image compositing.
    }
    else {
#define USE_ENVIRONMENT_MAP 1
#if USE_ENVIRONMENT_MAP
        if (environmentmap_dim->x != 0) {
            glm::vec3 rd = pathSegments[idx].ray.direction;
            float theta = acosf(rd.y), phi = atan2f(rd.z, rd.x);
            glm::vec2 dims = environmentmap_dim[0];

            float u = (phi + PI) * INV_2PI;
            float v = theta * INV_PI;
            int tex_x_idx = glm::fract(u) * dims.x;
            int tex_y_idx = glm::fract(v) * dims.y;
            int tex_1d_idx = tex_y_idx * dims.x + tex_x_idx;
            pathSegments[idx].color *= glm::vec3(environmentmap_data[tex_1d_idx]);
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
#else
        pathSegments[idx].color = glm::vec3(0.0f);
#endif
        pathSegments[idx].remainingBounces = 0;

    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

__global__ void normalizeOIDNBuffers(int iter_count, 
                                     int pixelcount, 
                                     glm::vec3* dev_oidn_albedo, 
                                     glm::vec3* dev_oidn_normalized_albedo,
                                     glm::vec3* dev_oidn_normal,
                                     glm::vec3* dev_oidn_normalized_normal)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < pixelcount)
    {
        dev_oidn_normalized_albedo[idx] = dev_oidn_albedo[idx] / (float)iter_count;
        dev_oidn_normalized_normal[idx] = dev_oidn_normal[idx] / (float)iter_count;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths; //just the pixel count for now

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_triangles,
            hst_scene->triangle_count,
            dev_bvhnodes
        );
        checkCUDAError("compute intersections");
        hipDeviceSynchronize();
        depth++;

#define USE_MATERIAL_SORTING 1
#if USE_MATERIAL_SORTING
        thrust::device_ptr<ShadeableIntersection> dev_inters_to_sort(dev_intersections);
        thrust::device_ptr<PathSegment> dev_paths_to_sort(dev_paths); //values
        thrust::stable_sort_by_key(dev_inters_to_sort, dev_inters_to_sort + num_paths, dev_paths_to_sort, CompareMaterials());
#endif

        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadeMaterials << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            depth,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_tex_data,
            dev_bumpmap_data,
            dev_environmentmap_data,
            dev_environmentmap_dim,
            dev_tex_starts,
            dev_bump_starts,
            dev_tex_dims,
            dev_bump_dims,
            dev_oidn_albedo,
            dev_oidn_normal
            );

        //std::cout << "Before iter " << depth << " there are " << num_paths << " paths\n";

#define USE_STREAM_COMPACTION 1
#if USE_STREAM_COMPACTION
        thrust::device_ptr<PathSegment> dev_paths_to_compact(dev_paths);
        thrust::device_ptr<PathSegment> last_elt = thrust::stable_partition(thrust::device, dev_paths_to_compact, dev_paths_to_compact + num_paths, ShouldTerminate());
        num_paths = last_elt.get() - dev_paths;
#endif

        //std::cout << "After iter " << depth << " there are " << num_paths << " paths\n";
        
        iterationComplete = (depth >= traceDepth || num_paths == 0);

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

#define USE_OIDN_FOR_RENDER 0
#if USE_OIDN_FOR_RENDER

    normalizeOIDNBuffers << <numBlocksPixels, blockSize1d >> > (
        iter, 
        pixelcount, 
        dev_oidn_albedo, 
        dev_oidn_normalized_albedo, 
        dev_oidn_normal, 
        dev_oidn_normalized_normal
        );
    //rt = ray tracing filter
    const glm::ivec2& res = cam.resolution;
    oidn::FilterRef filter = oidn_device.newFilter("RT");
    filter.setImage("color", dev_image, oidn::Format::Float3, res.x, res.y);
    filter.setImage("albedo", dev_oidn_normalized_albedo, oidn::Format::Float3, res.x, res.y);
    filter.setImage("normal", dev_oidn_normalized_normal, oidn::Format::Float3, res.x, res.y);
    filter.setImage("output", dev_oidn_filtered_image, oidn::Format::Float3, res.x, res.y);
    filter.set("hdr", true);
    filter.commit();

    filter.execute();
    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_oidn_filtered_image);
#else

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
#endif

    checkCUDAError("pathtrace");
}

// Retrieve image from GPU
void updateSceneRender(glm::ivec2& dims) {
    int pixelcount = dims.x * dims.y;

#define USE_OIDN_FINAL_IMAGE 1
#if !USE_OIDN_FINAL_IMAGE
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
#else

    oidn::FilterRef filter = oidn_device.newFilter("RT");
    filter.setImage("color", dev_image, oidn::Format::Float3, dims.x, dims.y);
    filter.setImage("albedo", dev_oidn_normalized_albedo, oidn::Format::Float3, dims.x, dims.y);
    filter.setImage("normal", dev_oidn_normalized_normal, oidn::Format::Float3, dims.x, dims.y);
    filter.setImage("output", dev_oidn_filtered_image, oidn::Format::Float3, dims.x, dims.y);
    filter.set("hdr", true);
    filter.set("cleanAux", true);
    filter.commit();

    oidn::FilterRef albedoFilter = oidn_device.newFilter("RT");
    albedoFilter.setImage("albedo", dev_oidn_normalized_albedo, oidn::Format::Float3, dims.x, dims.y);
    albedoFilter.setImage("output", dev_oidn_normalized_albedo, oidn::Format::Float3, dims.x, dims.y);
    albedoFilter.commit();
    
    oidn::FilterRef normalFilter = oidn_device.newFilter("RT");
    normalFilter.setImage("normal", dev_oidn_normalized_normal, oidn::Format::Float3, dims.x, dims.y);
    normalFilter.setImage("output", dev_oidn_normalized_normal, oidn::Format::Float3, dims.y, dims.y);
    normalFilter.commit();

    albedoFilter.execute();
    normalFilter.execute();

    filter.execute();

    hipMemcpy(hst_scene->state.image.data(), dev_oidn_filtered_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
#endif
}
