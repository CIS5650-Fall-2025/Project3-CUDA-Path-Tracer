#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__host__ __device__ float meshRayIntersectionTest(
    Geom mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside,
    int vertexSize,
    Vertex* vertices)
{
    // Iterate through all triangles in this mesh
    float closestT = -1.0f;
    int bestTriIdx = -1;
    glm::vec2 v_range = mesh.vertex_indices;

    for (int i = v_range.x; i <= v_range.y; i += 3) {
        // Fetch triangle vertices
        glm::vec3 a = vertices[i].position;
        glm::vec3 b = vertices[i + 1].position;
        glm::vec3 c = vertices[i + 2].position;

        // Möller–Trumbore intersection
        glm::vec3 ab = b - a;
        glm::vec3 ac = c - a;
        glm::vec3 pvec = cross(r.direction, ac);
        float det = dot(ab, pvec);

        if (fabs(det) < EPSILON) continue;

        float invDet = 1.0f / det;
        glm::vec3 tvec = r.origin - a;
        float u = dot(tvec, pvec) * invDet;
        if (u < 0.0f || u > 1.0f) continue;

        glm::vec3 qvec = cross(tvec, ab);
        float v = dot(r.direction, qvec) * invDet;
        if (v < 0.0f || u + v > 1.0f) continue;

        float t = dot(ac, qvec) * invDet;
        if (t > EPSILON && (closestT == -1.0f || t < closestT)) {
            closestT = t;
            bestTriIdx = i;
        }
    }

    if (closestT < 0.0f) return -1.0f;

    intersectionPoint = r.origin + closestT * r.direction;
    glm::vec3 bary = barycentricInterp(
        vertices[bestTriIdx].position,
        vertices[bestTriIdx + 1].position,
        vertices[bestTriIdx + 2].position,
        intersectionPoint);

    // Interpolate normals
    normal =
        vertices[bestTriIdx].normal * bary.x +
        vertices[bestTriIdx + 1].normal * bary.y +
        vertices[bestTriIdx + 2].normal * bary.z;

    outside = dot(normalize(normal), normalize(r.direction)) <= 0.f;
    return closestT;
}

__host__ __device__ glm::vec3 barycentricInterp(const glm::vec3& a, const glm::vec3& b, const glm::vec3& c, const glm::vec3& p) {
    float areaABC = triArea(a, b, c);
    float areaPBC = triArea(p, b, c);
    float areaPCA = triArea(p, c, a);
    float areaPAB = triArea(p, a, b);
    return glm::vec3(areaPBC / areaABC, areaPCA / areaABC, areaPAB / areaABC);
}

__host__ __device__ float triArea(const glm::vec3& x, const glm::vec3& y, const glm::vec3& z) {
    return 0.5f * glm::length(glm::cross(z - y, x - y));
}
