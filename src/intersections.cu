#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = glm::min(t1, t2);
        outside = true;
    }
    else
    {
        t = glm::max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}


__host__ __device__ bool triangleIntersectionTest(
    Ray r,
    Triangle triangle,
    float& t,
    glm::vec3& baryPosition)
{
    // M�ller�Trumbore intersection algorithm
    const float epsilon = 1e-8f;
    glm::vec3 vertex0 = triangle.v0;
    glm::vec3 vertex1 = triangle.v1;
    glm::vec3 vertex2 = triangle.v2;

    glm::vec3 edge1 = vertex1 - vertex0;
    glm::vec3 edge2 = vertex2 - vertex0;
    glm::vec3 h = glm::cross(r.direction, edge2);
    float a = glm::dot(edge1, h);

    if (fabs(a) < epsilon)
        return false; // Ray is parallel to triangle

    float f = 1.0f / a;
    glm::vec3 s = r.origin - vertex0;
    float u = f * glm::dot(s, h);
    if (u < 0.0f || u > 1.0f)
        return false;

    glm::vec3 q = glm::cross(s, edge1);
    float v = f * glm::dot(r.direction, q);
    if (v < 0.0f || u + v > 1.0f)
        return false;

    float temp_t = f * glm::dot(edge2, q);
    if (temp_t > epsilon)
    {
        t = temp_t;
        baryPosition = glm::vec3(u, v, 0.0f);
        return true;
    }
    else
        return false;
}

__device__ void swap(float& a, float& b) {
    float temp = a;
    a = b;
    b = temp;
};

__device__ bool aabbIntersectionTest(const Ray& ray, const glm::vec3& min, const glm::vec3& max, float& t)
{
    float tmin = (min.x - ray.origin.x) / ray.direction.x;
    float tmax = (max.x - ray.origin.x) / ray.direction.x;

    // Swap tmin and tmax if needed
    if (tmin > tmax) swap(tmin, tmax);

    float tymin = (min.y - ray.origin.y) / ray.direction.y;
    float tymax = (max.y - ray.origin.y) / ray.direction.y;

    // Swap tymin and tymax if needed
    if (tymin > tymax) swap(tymin, tymax);

    if ((tmin > tymax) || (tymin > tmax))
        return false;

    if (tymin > tmin)
        tmin = tymin;

    if (tymax < tmax)
        tmax = tymax;

    float tzmin = (min.z - ray.origin.z) / ray.direction.z;
    float tzmax = (max.z - ray.origin.z) / ray.direction.z;

    // Swap tzmin and tzmax if needed
    if (tzmin > tzmax) swap(tzmin, tzmax);

    if ((tmin > tzmax) || (tzmin > tmax))
        return false;

    if (tzmin > tmin)
        tmin = tzmin;

    if (tzmax < tmax)
        tmax = tzmax;

    // Check if the ray starts inside the box
    bool inside = (ray.origin.x > min.x && ray.origin.x < max.x) &&
        (ray.origin.y > min.y && ray.origin.y < max.y) &&
        (ray.origin.z > min.z && ray.origin.z < max.z);

    // If the ray starts inside the box, we need to use tmax as the exit point
    if (inside)
    {
        t = tmax;
    }
    else
    {
        t = tmin;
    }

    return t >= 0.0f;
}
