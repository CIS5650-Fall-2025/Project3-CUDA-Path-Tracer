#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}



// From https://fileadmin.cs.lth.se/cs/Personal/Tomas_Akenine-Moller/raytri/ (Algorithm 2)
__host__ __device__ bool intersectTriangle(
    const glm::vec3& orig,
    const glm::vec3& dir,
    const glm::vec3& v0,
    const glm::vec3& v1,
    const glm::vec3& v2,
    float& t,
	glm::vec2& triPos)
{
    //double edge1[3], edge2[3], tvec[3], pvec[3], qvec[3];
    //double det, inv_det;

    const float eps = 0.000001f;

    /* find vectors for two edges sharing vert0 */
    glm::vec3 edge1 = v1 - v0;
    glm::vec3 edge2 = v2 - v0;

    /* begin calculating determinant - also used to calculate U parameter */
	glm::vec3 pvec = glm::cross(dir, edge2);

    /* if determinant is near zero, ray lies in plane of triangle */
	float det = glm::dot(edge1, pvec);

    /* calculate distance from vert0 to ray origin */
	glm::vec3 tvec = orig - v0;
    float inv_det = 1.0f / det;

    glm::vec3 qvec;
    if (det > eps)
    {
        /* calculate U parameter and test bounds */
		triPos.x = glm::dot(tvec, pvec);
        if (triPos.x < 0.0 || triPos.x > det)
            return false;

        /* prepare to test V parameter */
		qvec = glm::cross(tvec, edge1);

        /* calculate V parameter and test bounds */
		triPos.y = glm::dot(dir, qvec);
		if (triPos.y < 0.0 || triPos.x + triPos.y > det)
			return false;


    }
    else if (det < -eps)
    {
        /* calculate U parameter and test bounds */
		triPos.x = glm::dot(tvec, pvec);
		if (triPos.x > 0.0 || triPos.x < det)
			return false;

        /* prepare to test V parameter */
		qvec = glm::cross(tvec, edge1);

        /* calculate V parameter and test bounds */
		triPos.y = glm::dot(dir, qvec);
		if (triPos.y > 0.0 || triPos.x + triPos.y < det)
			return false;
    }
    else return false;  /* ray is parallell to the plane of the triangle */

    /* calculate t, ray intersects triangle */
	t = glm::dot(edge2, qvec) * inv_det;
	triPos *= inv_det;
    return true;
}

__host__ __device__ float meshIntersectionTest(
    Geom mesh,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside,
    Triangle* triangles) {

    Ray rt;
    rt.origin = multiplyMV(mesh.inverseTransform, glm::vec4(r.origin, 1.0f));
    rt.direction = glm::normalize(multiplyMV(mesh.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float t;
    float tmin = FLT_MAX;
    glm::vec2 triPos;
	glm::vec2 minTriPos;
    int minIdx;

    int maxIndex = mesh.triangleStartIdx + mesh.triangleCount;
    for (int i = mesh.triangleStartIdx; i < maxIndex; ++i)
    {
        const glm::vec3 v1 = triangles[i].v1;
        const glm::vec3 v2 = triangles[i].v2;
        const glm::vec3 v3 = triangles[i].v3;
        if (!intersectTriangle(
            rt.origin,
            rt.direction,
            v1,
            v2,
            v3,
            t, triPos))
        {
            continue;
        }

        if (t < tmin)
        {
            tmin = t;
            minTriPos = triPos;
            minIdx = i;
        }
    }

    if (tmin == FLT_MAX) return -1;

    glm::vec3 intersectionPointLocal = getPointOnRay(rt, tmin);
    glm::vec3 normalLocal = glm::normalize(glm::cross(triangles[minIdx].v2 - triangles[minIdx].v1, triangles[minIdx].v3 - triangles[minIdx].v1));

    intersectionPoint = multiplyMV(mesh.transform, glm::vec4(intersectionPointLocal, 1.f));
    normal = glm::normalize(multiplyMV(mesh.invTranspose, glm::vec4(normalLocal, 0.f)));

    outside = glm::dot(normal, r.direction) < 0;

    return t;
}