#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__host__ __device__ float meshIntersectionTest(
    const Mesh &mesh,
    const Ray &r,
    glm::vec3 &intersection_point,
    glm::vec3 &normal,
    bool &outside    
) {
    glm::vec3 const ro = multiplyMV(mesh.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 const rd = glm::normalize(multiplyMV(mesh.inverseTransform, glm::vec4(r.direction, 0.0f)));
    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float t = -1;

    for (int i = 0; i < mesh.num_indices; i += 3) {
        glm::vec3 triangle[3] = {
            mesh.vertices[mesh.indices[i]],
            mesh.vertices[mesh.indices[i + 1]],
            mesh.vertices[mesh.indices[i + 2]],
        };

        glm::vec3 barycentric;
        if (glm::intersectRayTriangle(ro, rd, triangle[0], triangle[1], triangle[2], barycentric)) {
            const auto ab = triangle[1] - triangle[0];
            const auto ac = triangle[2] - triangle[0];
            glm::vec3 intersection = triangle[0] + barycentric.x * ab + barycentric.y * ac;

            float current_t = glm::length(intersection - rt.origin);
            if (t < 0 || current_t < t) {
                t = current_t;
                intersection_point = intersection;
                normal = glm::normalize(glm::cross(ab, ac));
            }
        }
    }

    if (t < 0) {
        return -1;
    }

    intersection_point = multiplyMV(mesh.transform, glm::vec4{intersection_point, 1.f});
    normal = glm::normalize(multiplyMV(mesh.invTranspose, glm::vec4(normal, 0.f)));
    outside = glm::dot(normal, rt.direction) < 0;
    if (!outside) {
        normal = -normal;
    }

    return glm::length(r.origin - intersection_point);
}