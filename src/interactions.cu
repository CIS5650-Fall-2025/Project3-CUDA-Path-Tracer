#include "hip/hip_runtime.h"
#include "interactions.h"

__device__ bool Refract(const glm::vec3& wi, const glm::vec3& n, float eta, glm::vec3& w_r) {
    // Compute cos theta using Snell's law
    float cosThetaI = dot(n, wi);
    float sin2ThetaI = glm::max(float(0), float(1 - cosThetaI * cosThetaI));
    float sin2ThetaT = eta * eta * sin2ThetaI;

    // Handle total internal reflection for transmission
    if (sin2ThetaT >= 1) return false;
    float cosThetaT = sqrt(1 - sin2ThetaT);
    w_r = eta * -wi + (eta * cosThetaI - cosThetaT) * n;
    return true;
}

__device__ void squareToDiskConcentric(const glm::vec2 xi, glm::vec3& wi)
{
    //Remap to [-1, 1], [-1, 1]
    glm::vec2 offset = 2.f * xi - glm::vec2(1, 1);
    if (offset.x == 0 && offset.y == 0)
    {
        //Handle base case
        wi = glm::vec3(0);
    }

    // Apply concentric mapping to point
    float theta, r;
    if (abs(offset.x) > abs(offset.y)) {
        r = offset.x;
        theta = PI_OVER_FOUR * (offset.y / offset.x);
    }
    else {
        r = offset.y;
        theta = PI_OVER_TWO - PI_OVER_FOUR * (offset.x / offset.y);
    }
    wi = r * glm::vec3(cos(theta), sin(theta), 0);
}

__device__ void squareToHemisphereCosine(const glm::vec2 xi, glm::vec3 &wi) {
    squareToDiskConcentric(xi, wi);
    //Extrapolate z using x, y coords of the point, uniformly sampled at the base of the hemisphere!
    float z = sqrt(glm::max(0.f, 1.f - wi.x * wi.x - wi.y * wi.y));
    wi.z = z;
}

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

/**
* The function FrDielectric() computes the Fresnel reflection formula for dielectric materials and unpolarized light.
* REMEMBER: Reflection != Refraction!
**/
__device__ float FresnelDielectricEval(float cI)
{
    float etai = 1.;
    float etat = 1.55;
    float cosThetaI = glm::clamp(cI, -1.f, 1.f);

    // Potentially swap 
    bool entering = cosThetaI > 0.f;
    if (!entering) {
        float temp = etai;
        etai = etat;
        etat = temp;
        cosThetaI = abs(cosThetaI);
    }

    // Snells Law
    float eta = etai / etat;
    float sint = eta * sqrtf(glm::max(0.0f, 1.0f - cosThetaI * cosThetaI));

    // TIR
    if (sint >= 1.0f) {
        return 1.0f; // 100% reflection
    }

    // Calculate cos of the transmission angle
    float cost = sqrtf(glm::max(0.0f, 1.0f - sint * sint));

    // Schlick approximation
    float Rs = ((etat * cosThetaI) - (etai * cost)) / ((etat * cosThetaI) + (etai * cost));
    float Rp = ((etai * cosThetaI) - (etat * cost)) / ((etai * cosThetaI) + (etat * cost));
    return (Rs * Rs + Rp * Rp) * 0.5f;
}
__device__ void sample_f_diamond(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    float r = u01(rng);
    pdf = 1;

    glm::vec3 woWOut = pathSegment.ray.direction;

    float cosi = glm::dot(woWOut, normal);

    float fresnelReflectance = FresnelDielectricEval(cosi);

    if (m.roughness <= 0) {
        sample_f_diamond_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }
    else if (m.roughness > 1) {
        sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }


    if (r < m.roughness) {
        sample_f_diamond_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        f /= m.roughness;
    }
    else {
        sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        f *= (1.f - fresnelReflectance) / (1 - m.roughness);
    }
}


__device__ void sample_f_glass(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    float r = u01(rng);
    pdf = 1;

    glm::vec3 woWOut = -pathSegment.ray.direction;

    float cosi = glm::dot(woWOut, normal);

    float fresnelReflectance = FresnelDielectricEval(cosi);

    if (m.roughness <= 0) {
        sample_f_specular_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }
    else if (m.roughness > 1) {
        sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }


    if (r < m.roughness) {
        sample_f_specular_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        f /= m.roughness;

        
        f *= fresnelReflectance;
    }
    else {
        sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        f /= (1 - m.roughness);
        f *= (1 - fresnelReflectance);
    }
}

__device__ void sample_f_diamond_refl(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{

    glm::vec3 wi = glm::vec3(-woOut.x, -woOut.y, woOut.z);
    if (dot(woOut, normal) > 0) {
        wi = -wi;
    }
    pathSegment.ray.direction = wi;
    pdf = 1;
    glm::vec3 col = m.color;
    if (useTexCol) {
        col = texCol;
    }
    f = col / AbsCosTheta(wi);
}

__device__ void sample_f_specular_refl(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{

    glm::vec3 wi = glm::vec3(-woOut.x, -woOut.y, woOut.z);
    //if (dot(pathSegment.ray.direction, normal) > 0) {
    //    wi = -wi;
    //}
    pathSegment.ray.direction = wi;
    pdf = 1;
    glm::vec3 col = m.color;
    if (useTexCol) {
        col = texCol;
    }
    f = col / AbsCosTheta(wi);
}


__device__ void sample_f_specular_trans(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    pdf = 1;
    // IOR of glass! (refraction based on Snell's law depends on the IOR of mediums. In this case, we have air and glass.)
    float etaA = 1.f;
    float etaB = 1.55f;

    // Potentially swap due to whether we are entering or exiting the glass
    bool entering = CosTheta(woOut) > 0.f;
    float etaI = entering ? etaA : etaB; // incident index
    float etaT = entering ? etaB : etaA; // transmitted index

    // compute ray direction for specular trans
    glm::vec3 wi;
    glm::vec3 wo_local = woOut;

    if (!Refract(wo_local, Faceforward(glm::vec3(0, 0, 1), wo_local), etaI / etaT, wi)) {
        pdf = 0;
        return;
    }

    pathSegment.ray.direction = wi;

    //f = glm::vec3(1, 1, 1);
    glm::vec3 col = m.color;
    if (useTexCol) {
        col = texCol;
    }
    f = col / AbsCosTheta(wi);
}

__device__ float Lambda(glm::vec3 w, float roughness) {
    float absTanTheta = abs(TanTheta(w));
    if (isinf(absTanTheta)) return 0.;

    // Compute alpha for direction w
    float alpha =
        sqrt(Cos2Phi(w) * roughness * roughness + Sin2Phi(w) * roughness * roughness);
    float alpha2Tan2Theta = (roughness * absTanTheta) * (roughness * absTanTheta);
    return (-1 + sqrt(1.f + alpha2Tan2Theta)) / 2;
}


__device__ float TrowbridgeReitzG(glm::vec3 wo, glm::vec3 wi, float roughness) {
    return 1 / (1 + Lambda(wo, roughness) + Lambda(wi, roughness));
}

__device__ float TrowbridgeReitzD(glm::vec3 wh, float roughness) {
    float tan2Theta = Tan2Theta(wh);
    if (isinf(tan2Theta)) return 0.f;

    float cos4Theta = Cos2Theta(wh) * Cos2Theta(wh);

    float e = (Cos2Phi(wh) / (roughness * roughness) + Sin2Phi(wh) / (roughness * roughness)) * tan2Theta;
    return 1 / (PI * roughness * roughness * cos4Theta * (1 + e) * (1 + e));
}

__device__ float TrowbridgeReitzPdf(glm::vec3 wh, float roughness) {
    return TrowbridgeReitzD(wh, roughness) * AbsCosTheta(wh);
}

__device__ glm::vec3 sample_wh(glm::vec3 wo, glm::vec2 xi, float roughness) {
    glm::vec3 wh;

    float cosTheta = 0;
    float phi = TWO_PI * xi[1];

    // isotropic microfacet materials only
    float tanTheta2 = roughness * roughness * xi[0] / (1.0f - xi[0]);
    cosTheta = 1 / sqrt(1 + tanTheta2);

    float sinTheta =
        sqrt(glm::max(0.f, 1.f - cosTheta * cosTheta));

    wh = glm::vec3(sinTheta * cos(phi), sinTheta * sin(phi), cosTheta);

    if (!SameHemisphere(wo, wh))
    {
        wh = -wh;
    }

    return wh;
}

__device__ glm::vec3 f_microfacet_refl(glm::vec3 col, glm::vec3 woOut, glm::vec3 wi, float roughness){
    float cosThetaO = AbsCosTheta(woOut);
    float cosThetaI = AbsCosTheta(wi);
    glm::vec3 wh = wi + woOut;
    // Handle degenerate cases for microfacet reflection
    if (cosThetaI == 0 || cosThetaO == 0) return glm::vec3(0.f);
    if (wh.x == 0 && wh.y == 0 && wh.z == 0) return glm::vec3(0.f);
    wh = normalize(wh);

    glm::vec3 F = glm::vec3(1.);
    float D = TrowbridgeReitzD(wh, roughness);
    float G = TrowbridgeReitzG(woOut, wi, roughness);
    return col * D * G * F / (4 * cosThetaI * cosThetaO);
}

__device__ void sample_f_microfacet_refl(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    if (woOut.z == 0) {
        f = glm::vec3(0);
        pdf = 0;
        return;
    }

    //We need to sample the microfacet normal!
    thrust::uniform_real_distribution<float> u01(0, 1);
    const glm::vec2 xi = glm::vec2(u01(rng), u01(rng));
    glm::vec3 wh = sample_wh(woOut, xi, m.roughness);

    glm::vec3 wi = glm::reflect(-woOut, wh);

    if (!SameHemisphere(woOut, wi)) {
        f = glm::vec3(0);
        pdf = 0;
        return;
    }
    glm::vec3 col = m.color;
    if (useTexCol) {
        col = texCol;
    }

    pdf = TrowbridgeReitzPdf(wh, m.roughness) / (4 * dot(woOut, wh));
    f = f_microfacet_refl(col, woOut, wi, m.roughness);
    pathSegment.ray.direction = wi;
}


__device__ void sample_f_ceramic_refl(
    PathSegment& pathSegment,
    const glm::vec3& woOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);
    float r = u01(rng);
    pdf = 1;

    glm::vec3 woWOut = pathSegment.ray.direction;

    float cosi = glm::dot(woWOut, normal);

    float fresnelReflectance = FresnelDielectricEval(cosi);

    if (m.roughness <= 0) {
        sample_f_diamond_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }
    else if (m.roughness > 1) {
        sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        return;
    }

    if (r < m.roughness) {
        sample_f_specular_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
        //f /= m.roughness;
    }
    else {
        sample_f_diffuse(pathSegment, pdf, f, normal, m, texCol, useTexCol, rng);
        //f *= (1.f - fresnelReflectance) / (1 - m.roughness);
    }
}

__device__ void f_diffuse(
    glm::vec3& f,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol)
{
    glm::vec3 col = m.color;
    if (useTexCol) {
        col = texCol;
    }
    f = INV_PI * col;
}

__device__ void pdf_diffuse(
    float& pdf, const glm::vec3& wi)
{
    pdf = INV_PI * AbsCosTheta(wi);
}

__device__ void sample_f_diffuse(
    PathSegment& pathSegment,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    bool useTexCol,
    thrust::default_random_engine& rng)
{
    //0. rng gen
    thrust::uniform_real_distribution<float> u01(0, 1);
    const glm::vec2 xi = glm::vec2(u01(rng), u01(rng));
    //1. Generate wi (local space)
    glm::vec3 wi = glm::vec3(0);
    squareToHemisphereCosine(xi, wi);
    //2. Find f
    f_diffuse(f, m, texCol, useTexCol);

    //3. Find pdf
    pdf_diffuse(pdf, wi);
    //4. update wi
    pathSegment.ray.direction = wi;
}
__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 normal,
    thrust::default_random_engine& rng)
{
    //Update ray in pathSegment
    pathSegment.ray.direction = calculateRandomDirectionInHemisphere(normal, rng);
}

__device__ void sample_f(
    PathSegment& pathSegment,
    const glm::vec3& woWOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    const bool useTexCol,
    thrust::default_random_engine& rng)
{
    glm::vec3 woOut = WorldToLocal(normal) * woWOut;

    switch (m.type) {
        case DIFFUSE_REFL:
            sample_f_diffuse(pathSegment, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case SPEC_REFL:
            sample_f_specular_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case SPEC_TRANS:
            sample_f_specular_trans(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case SPEC_GLASS:
            sample_f_glass(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case MICROFACET_REFL:
            sample_f_microfacet_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case DIAMOND:
            sample_f_diamond(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case CERAMIC:
            sample_f_ceramic_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            break;
        case MATTEBLACK:
            sample_f_diffuse(pathSegment, pdf, f, normal, m, texCol, useTexCol, rng);
            f = glm::vec3(0.01, 0.01, 0.01);
            break;
        default:
            sample_f_diffuse(pathSegment, pdf, f, normal, m, texCol, useTexCol, rng);
    }

    pathSegment.ray.direction = LocalToWorld(normal) * pathSegment.ray.direction;
}

__device__ void f(
    const glm::vec3& woWOut,
    const glm::vec3& wiWOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    const bool useTexCol,
    thrust::default_random_engine& rng)
{
    glm::vec3 woOut = WorldToLocal(normal) * woWOut;
    glm::vec3 wiOut = WorldToLocal(normal) * wiWOut;

    switch (m.type) {
        case DIFFUSE_REFL:
            f_diffuse(f, m, texCol, useTexCol);
            break;
        case SPEC_REFL:
        case SPEC_TRANS:
        case SPEC_GLASS:
        case DIAMOND:
            f = glm::vec3(0);
            break;
        case MICROFACET_REFL:
            glm::vec3 col = m.color;
            if (useTexCol) {
                col = texCol;
            }
            f = f_microfacet_refl(col, woOut, wiOut, m.roughness);
            break;
        case CERAMIC:
            //sample_f_ceramic_refl(pathSegment, woOut, pdf, f, normal, m, texCol, useTexCol, rng);
            f_diffuse(f, m, texCol, useTexCol);
            break;
        case MATTEBLACK:
            f = glm::vec3(0.01, 0.01, 0.01);
            break;
        default:
            //sample_f_diffuse(pathSegment, pdf, f, normal, m, texCol, useTexCol, rng);
            f = glm::vec3(1, 0, 1);
    }
}

__device__ void pdf(
    const glm::vec3& woWOut,
    const glm::vec3& wiWOut,
    float& pdf,
    glm::vec3& f,
    glm::vec3 normal,
    const Material& m,
    const glm::vec3 texCol,
    const bool useTexCol,
    thrust::default_random_engine& rng)
{
    glm::vec3 woOut = WorldToLocal(normal) * woWOut;
    glm::vec3 wiOut = WorldToLocal(normal) * wiWOut;

    if (woOut.z == 0) {
        pdf = 0;
        return;
    }

    switch (m.type) {
    case DIFFUSE_REFL:
    case MATTEBLACK:
        pdf_diffuse(pdf, wiOut);
        break;
    case SPEC_REFL:
    case SPEC_TRANS:
    case SPEC_GLASS:
    case DIAMOND:
        pdf = 0;
        break;
    case MICROFACET_REFL:
        glm::vec3 wh = normalize(woOut + wiOut);
        pdf = TrowbridgeReitzPdf(wh, m.roughness) / (4 * dot(woOut, wh));
        break;
    case CERAMIC:
        pdf_diffuse(pdf, wiOut);
        break;
    default:
        pdf_diffuse(pdf, wiOut);
    }
}