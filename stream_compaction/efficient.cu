#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Helper functions
        // Up-Sweep - parallel reduction
        __global__ void upSweep(int n, int d, int* data)
        {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            int index = idx << (d + 1);

            if (index + (1 << (d + 1)) - 1 < n) 
            {
                // From the slides:
                // x[k + 2d+1 � 1] += x[k + 2d � 1];
                data[index + (1 << (d + 1)) - 1] += data[index + (1 << d) - 1];
            }
        }

        // Down-Sweep - traverse back down the tree using partial sums to build the scan in place.
        __global__ void downSweep(int n, int d, int* data)
        {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            int index = idx << (d + 1);

            if (index + (1 << (d + 1)) - 1 < n) 
            {
                // Save left child
                int temp = data[index + (1 << d) - 1];

                // Set left child to this node�s value
                data[index + (1 << d) - 1] = data[index + (1 << (d + 1)) - 1];

                // Set right child to old left value + this node�s value
                data[index + (1 << (d + 1)) - 1] += temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // DONE
            // This approach uses balanced trees to avoid the extra factor of log2n work
            // performed by the naive algorithm.
            // GPU Gems 3, Chapter 39.2.2

            int* dev_idata;
            // Calculate number of levels needed for the scan
            // ilog2ceil(x): computes the ceiling of log2(x), as an integer.
            int numLevels = ilog2ceil(n);
            // Calculate the power of 2 number of levels
            int numLevelsPow2 = 1 << numLevels;
            
            // Memory allocation
            hipMalloc((void**)&dev_idata, numLevelsPow2 * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed!");

            int padding = numLevelsPow2 - n;
            if (padding >= 0)
            {
                hipMemset(&dev_idata[n], 0, padding * sizeof(int));
                checkCUDAError("hipMemset dev_idata failed!");
            }
            
            timer().startGpuTimer();

            //================================================================================
            // Part 1 - Upsweep phase
            //================================================================================
            for (int offset = 0; offset < numLevels - 1; offset++) 
            {
                // Calculate the number of blocks
                int numBlocks = (numLevelsPow2 / (1 << (offset + 1)) + blockSize - 1) / blockSize;
                
                // Perform the upsweep phase
                upSweep << <numBlocks, blockSize >> > (numLevelsPow2, offset, dev_idata);
                checkCUDAError("upSweep kernel failed!");

                // Sync before proceeding to the next iteration
                hipDeviceSynchronize();
            }

            // Need to set the last element to 0 before starting the down sweep phase 
            hipMemset(dev_idata + numLevelsPow2 - 1, 0, sizeof(int));
            checkCUDAError("hipMemset dev_idata failed!");

            //================================================================================
            // Part 2 - Downsweep phase
            //================================================================================
            for (int offset = numLevels - 1; offset >= 0; offset--) {
                // Calculate the number of blocks
                int numBlocks = (numLevelsPow2 / (1 << (offset + 1)) + blockSize - 1) / blockSize;
                
                // Perform the downsweep phase
                downSweep << <numBlocks, blockSize >> > (numLevelsPow2, offset, dev_idata);
                checkCUDAError("downSweep kernel failed!");

                // Sync before proceeding to the next iteration
                hipDeviceSynchronize();
            }

            timer().endGpuTimer();

            // Copy the results back to the host
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy device to host (dev_idata to odata) failed!");

            // Free device memory
            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // This stream compaction method will remove 0s from an array of ints.
            // Initialize necessary buffers
            int* dev_idata;
            int* dev_odata;
            int* dev_booleans;
            int* dev_indices;

            // ilog2ceil(x): computes the ceiling of log2(x), as an integer.
            int numLevels = ilog2ceil(n);
            size_t paddedSize = (size_t) 1 << numLevels;

            // Allocate memory for device arrays, copy input data to device
            hipMalloc((void**)&dev_idata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");
            hipMalloc((void**)&dev_odata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_booleans, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_booleans failed");
            hipMalloc((void**)&dev_indices, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed");
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed");

            if (paddedSize > n) {
                hipMemset(dev_idata + n, 0, (paddedSize - n) * sizeof(int));
                checkCUDAError("hipMemset dev_idata failed!");
            }

            dim3 fullBlocksPerGrid((paddedSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            // Map to boolean
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (paddedSize, dev_booleans, dev_idata);
            checkCUDAError("kernMapToBoolean failed!");

            // Perform scan on the boolean array
            hipMemcpy(dev_indices, dev_booleans, sizeof(int) * paddedSize, hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy from dev_booleans to dev_indices (Device To Device) failed!");

            //================================================================================
            // Part 1 - Upsweep phase
            //================================================================================
            for (int offset = 0; offset < numLevels - 1; offset++) {
                
                // Calculate necessary number of blocks
                int numBlocks = (paddedSize / (1 << (offset + 1)) + blockSize - 1) / blockSize;
                
                if (numBlocks > 0) 
                {
                    upSweep << <numBlocks, blockSize >> > (paddedSize, offset, dev_indices);
                    checkCUDAError("upSweep kernel failed!");
                    hipDeviceSynchronize();
                }
            }

            // Need to set the last element to 0 before starting the down sweep phase 
            hipMemset(dev_indices + paddedSize - 1, 0, sizeof(int));
            checkCUDAError("hipMemset failed!");

            //================================================================================
            // Part 2 - Downsweep phase
            //================================================================================
            for (int offset = numLevels - 1; offset >= 0; offset--) {
                int numBlocks = (paddedSize / (1 << (offset + 1)) + blockSize - 1) / blockSize;
                if (numBlocks > 0) 
                {
                    downSweep << <numBlocks, blockSize >> > (paddedSize, offset, dev_indices);
                    checkCUDAError("downSweep kernel failed!");
                    hipDeviceSynchronize();
                }
            }

            //================================================================================
            // Part 3: Scatter
            //================================================================================
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (paddedSize, dev_odata, dev_idata, dev_booleans, dev_indices);
            checkCUDAError("kernScatter failed!");

            timer().endGpuTimer();

            //================================================================================
            // Part 4: Copy results and free memory
            //================================================================================
            int numRemaining;
            hipMemcpy(&numRemaining, dev_indices + paddedSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy numRemaining failed!");

            // Copy result from device to host
            hipMemcpy(odata, dev_odata, sizeof(int) * numRemaining, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");

            // Free memory of temp buffers
            hipFree(dev_booleans);
            hipFree(dev_indices);
            hipFree(dev_idata);
            hipFree(dev_odata);

            return numRemaining;
        }
    }
}
